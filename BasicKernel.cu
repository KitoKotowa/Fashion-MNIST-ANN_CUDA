#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define BLOCK_SIZE 256 
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", "; \
        cerr << "code: " << error << ", reason: " << hipGetErrorString(error) << endl; \
        throw runtime_error(hipGetErrorString(error)); \
    } \
}

class NeuralNetworkCUDA {
private:
    const int input_size = 784;
    const int hidden1_size = 128;
    const int hidden2_size = 128; 
    const int output_size = 10;
    const float learning_rate = 0.01;

    float *d_W1, *d_W2, *d_W3;
    float *d_b1, *d_b2, *d_b3;
    float *d_input, *d_z1, *d_a1, *d_z2, *d_a2, *d_z3, *d_output;
    float *d_target, *d_dZ3, *d_dZ2, *d_dZ1;

    vector<float> h_W1, h_W2, h_W3;
    vector<float> h_b1, h_b2, h_b3;

    void allocate_memory() {
        // Weights and biases
        hipMalloc(&d_W1, hidden1_size * input_size * sizeof(float));
        hipMalloc(&d_W2, hidden2_size * hidden1_size * sizeof(float));
        hipMalloc(&d_W3, output_size * hidden2_size * sizeof(float));
        
        hipMalloc(&d_b1, hidden1_size * sizeof(float));   
        hipMalloc(&d_b2, hidden2_size * sizeof(float));
        hipMalloc(&d_b3, output_size * sizeof(float));
    }

    void initialize_weights() {
        random_device rd;
        mt19937 gen(rd());
        normal_distribution<float> d(0, 0.01);

        // Xavier/Glorot initialization
        h_W1.resize(hidden1_size * input_size);
        h_W2.resize(hidden2_size * hidden1_size);
        h_W3.resize(output_size * hidden2_size);
        
        h_b1.resize(hidden1_size, 0.0f);
        h_b2.resize(hidden2_size, 0.0f);
        h_b3.resize(output_size, 0.0f);

        for (int i = 0; i < h_W1.size(); ++i) {
            h_W1[i] = d(gen) * sqrt(2.0f / (input_size + hidden1_size));
        }

        for (int i = 0; i < h_W2.size(); ++i) {
            h_W2[i] = d(gen) * sqrt(2.0f / (hidden1_size + hidden2_size));
        }

        for (int i = 0; i < h_W3.size(); ++i) {
            h_W3[i] = d(gen) * sqrt(2.0f / (hidden2_size + output_size));
        }

        hipMemcpy(d_W1, h_W1.data(), h_W1.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W2, h_W2.data(), h_W2.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_W3, h_W3.data(), h_W3.size() * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(d_b1, h_b1.data(), h_b1.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b2, h_b2.data(), h_b2.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b3, h_b3.data(), h_b3.size() * sizeof(float), hipMemcpyHostToDevice);
    }
    void forward_propagation() {
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // First hidden layer
        // TODO: Implement CUDA kernels for matrix multiplication
        // Placeholder for CUDA gemm (general matrix multiplication)
        
        // ReLU activation
        int blocks1 = (hidden1_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        relu_kernel<<<blocks1, BLOCK_SIZE>>>(d_z1, d_a1, hidden1_size);
        
        // Second hidden layer
        // TODO: Similar matrix multiplication kernel
        
        // ReLU activation
        int blocks2 = (hidden2_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        relu_kernel<<<blocks2, BLOCK_SIZE>>>(d_z2, d_a2, hidden2_size);
        
        // Output layer
        // TODO: Final matrix multiplication
        
        // Softmax
        int blocks3 = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        softmax_kernel<<<blocks3, BLOCK_SIZE>>>(d_z3, d_output, output_size);
    }
public: 
    NeuralNetworkCUDA() {
        allocate_gpu_memory();
        initialize_weights();
    }

    ~NeuralNetworkCUDA() {
        // Free GPU memory
        hipFree(d_W1); hipFree(d_W2); hipFree(d_W3);
        hipFree(d_b1); hipFree(d_b2); hipFree(d_b3);
        hipFree(d_input); hipFree(d_z1); hipFree(d_a1);
        hipFree(d_z2); hipFree(d_a2); hipFree(d_z3);
        hipFree(d_output); hipFree(d_target);
        hipFree(d_dZ3); hipFree(d_dZ2); hipFree(d_dZ1);
    }

    void train(const vector<vector<float>>& training_data, 
               const vector<int>& labels, 
               int epochs = 5, 
               float learning_rate = 0.01) {
        //TODO: Train here
    
    }
};

__global__ void relu_kernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        output[idx] = max(0.0f, input[idx]);
    }
}

__global__ void relu_derivative_kernel(float* input, float* output, int size) { 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = input[idx] > 0.0f ? 1.0f : 0.0f;
    }
}

__global__ void softmax_kernel(float* input, float* output, int size) {
    __shared__ float max_val;
    __shared__ float sum;

    if (threadIdx.x == 0) {
        max_val = input[0];
        for  (int i = 1; i < size; ++size) {
            if (input[i] > max_val) {
                max_val = input[i];
            }
        }
    }

    __syncthreads();
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float exp_x = idx < size ? exp(input[idx] - max_val) : 0.0f;
    
    __shared__ float shared_mem[BLOCK_SIZE];
    shared_mem[threadIdx.x] = exp_x;

    for (int stride = blockDim.x / 2; stride > 0; stride >> 1) {
        __syncthreads();
        if (threadIdx.x < stride) {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + stride];
        }
    }
    
    if (threadIdx.x == 0) {
        sum = shared_mem[0];
    }

    __syncthreads();

    if (idx < size) {
        output[idx] = exp(input[idx] - max_val) / sum;
    }
}

bool loadFashionMNIST(vector<vector<float>>& images,
                      vector<int>& labels,
                      const string& image_file,
                      const string& label_file) {
    ifstream file_images(image_file, ios::binary); 
    ifstream file_labels(label_file, ios::binary);
    
    if(!file_images.is_open() || !file_labels.is_open()) {
        cout << "Error opening dataset files" << endl;
        return false;
    }
    
    // Read image file header
    int magic_number = 0, num_images = 0, num_rows = 0, num_cols = 0;
    file_images.read(reinterpret_cast<char*>(&magic_number), sizeof(magic_number));
    file_images.read(reinterpret_cast<char*>(&num_images), sizeof(num_images));
    file_images.read(reinterpret_cast<char*>(&num_rows), sizeof(num_rows));
    file_images.read(reinterpret_cast<char*>(&num_cols), sizeof(num_cols));
    
    // Read label file header
    file_labels.read(reinterpret_cast<char*>(&magic_number), sizeof(magic_number));
    file_labels.read(reinterpret_cast<char*>(&num_images), sizeof(num_images));
    
    // Convert from big-endian to little-endian
    magic_number = __builtin_bswap32(magic_number);
    num_images = __builtin_bswap32(num_images);
    num_rows = __builtin_bswap32(num_rows);
    num_cols = __builtin_bswap32(num_cols);
    
    // Read images and labels
    images.resize(num_images, vector<float>(num_rows * num_cols));
    labels.resize(num_images);
    
    for (int i = 0; i < num_images; ++i) {
        for (int j = 0; j < num_rows * num_cols; ++j) {
            unsigned char pixel = 0;
            file_images.read(reinterpret_cast<char*>(&pixel), sizeof(pixel));
            if (file_images.fail()) {
                cerr << "Error reading image data." << endl;
                return false;
            }
            images[i][j] = pixel / 255.0; // Normalize to [0, 1]
        }

        unsigned char label = 0;
        file_labels.read(reinterpret_cast<char*>(&label), sizeof(label));
        if (file_labels.fail()) {
            cerr << "Error reading label data." << endl;
            return false;
        }
        labels[i] = static_cast<int>(label);
    }
    
    return true;
}

void printTrainingData(const vector<vector<float>>& images,
                       const vector<int>& labels, int count) {
    for (int i = 0; i < count && i < images.size(); ++i) {
        cout.precision(2);
        cout << "Sample " << i + 1 << ":\n";
        cout << "Label: " << labels[i] << "\n";
        cout << "Image (Flattened):\n";
        for (int j = 0; j < images[i].size(); ++j) {
            cout << images[i][j] << " ";
            if ((j + 1) % 28 == 0) cout << "\n"; // Format into 28 pixels per row
        }
        cout << "\n" << string(40, '-') << "\n";
    }
}

int main() {
    vector<vector<float>> images;
    vector<int> labels;
    
    string image_file = "train-images-idx3-ubyte";
    string label_file = "train-labels-idx1-ubyte";
    
    if (!loadFashionMNIST(images, labels, image_file, label_file)) {
        cerr << "Failed to load Fashion MNIST dataset" << endl;
        return 1;
    }

    cout << "Loaded " << images.size() << " training samples.\n";
    cout << "Printing the first 5 samples:\n"; 
    printTrainingData(images, labels, 5);

    // Convert vector<vector<float>> to Eigen::MatrixXd
    MatrixXd training_data(images.size(), images[0].size());
    for (size_t i = 0; i < images.size(); i++) {
        for (size_t j = 0; j < images[i].size(); j++) {
            training_data(i, j) = images[i][j];
        }
    }

    // Convert labels to one-hot encoded matrix
    MatrixXd training_labels = MatrixXd::Zero(labels.size(), 10);
    for (size_t i = 0; i < labels.size(); i++) {
        training_labels(i, labels[i]) = 1.0;
    }

    // Create and train neural network
    NeuralNetworkCUDA nn;
    cout << "Training neural network...\n";
    nn.train(training_data, training_labels);
    cout << "Training complete!\n";
    
    return 0;
}